#include "hip/hip_runtime.h"
// Computes the L2-norm (dot product) between all rows of a matrix
// Matrix is stored in row-major format!
#include <cassert>
#include <iostream>
#include <vector>

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

// #include "cuda_kernels.h"
#include "host_utils.h"
#include "sort_vector.h"
#include "timer.h"
#include "utils.h"
#include "vanilla_attention_kernels.h"

#define FULL_MASK 0xffffffff
#define MAX_GRID_Y 65535

#define ERR_NE(X, Y)                                                           \
  do {                                                                         \
    if ((X) != (Y)) {                                                          \
      fprintf(stderr, "Error in %s at %s:%d\n", __func__, __FILE__, __LINE__); \
      exit(-1);                                                                \
    }                                                                          \
  } while (0)
#define CUDA_CALL(X) ERR_NE((X), hipSuccess)
#define CUSPARSE_CALL(X) ERR_NE((X), HIPSPARSE_STATUS_SUCCESS)

int main() {
  const int runs = 1;
#define FLOAT

#if defined(FLOAT)
  printf("Single precision\n");
  typedef float floatType;
// hipDataType cublasType = HIP_R_32F;
// hipblasComputeType_t cublasComputeType = HIPBLAS_COMPUTE_32F;
#undef TENSOR
#elif defined(TENSOR)
  printf("Tensor float precision\n");
  typedef float floatType;
// hipDataType cublasType = HIP_R_32F;
// hipblasComputeType_t cublasComputeType = HIPBLAS_COMPUTE_32F_FAST_TF32;
#undef TENSOR
#elif defined(DOUBLE)
  printf("Double precision\n");
  typedef double floatType;
// hipDataType cublasType = HIP_R_64F;
// hipblasComputeType_t cublasComputeType = HIPBLAS_COMPUTE_64F;
#undef DOUBLE
#endif

  CUDA_CHECK(hipSetDevice(1));
  const bool checkRMSE = true;
  const bool print_debug = false;

  const size_t rows = 10000;
  const size_t cols = 1024; // assumes that col < 1024 !
  assert((cols <= 1024) &&
         "Cols must be smaller than 1024 to fit in thread block");
  const double sparsity_density = 0.01;
  const size_t nnz = floor(rows * rows * sparsity_density);
  // size_t nnz = rows * rows;
  std::cout << "Rows: " << rows;
  std::cout << "; Cols: " << cols << '\n';
  std::cout << "nnz: " << nnz << '\n';
  const double numFlops = 2 * nnz * cols * 1e-9;
  const double numBytes = (2 * nnz * cols + 3 * nnz) * sizeof(floatType) * 1e-9;
  printf("Memory usage =         %f GB\n", numBytes);
  printf("Arithmetic Intensity:  %f FLOPs/Bytes\n", numFlops / numBytes);

  std::vector<size_t> A_rows_coo_h(nnz, 0);
  std::vector<size_t> A_cols_h(nnz, 0);

  for (size_t iter_idx = 0; iter_idx < nnz; iter_idx++) {
    A_rows_coo_h[iter_idx] = rand() % rows;
    A_cols_h[iter_idx] = rand() % rows;
  }

  // for (auto& e : A_rows_coo_h) std::cout << e << '\t';
  // std::cout << '\n';
  // for (auto& e : A_cols_h) std::cout << e << '\t' ;
  // std::cout << '\n';
  sort_vectors_by_row(A_rows_coo_h, A_cols_h);
  // for (auto& e : A_rows_coo_h) std::cout << e << '\t';
  // std::cout << '\n';
  // for (auto& e : A_cols_h) std::cout << e << '\t';
  // std::cout << '\n';

  auto A_rows_csr_h = coo_to_crs(A_rows_coo_h, rows);

  // This loop prints duplicates
  // for (size_t row_ptr = 0; row_ptr < A_rows_csr_h.size() - 1; row_ptr++) {
  //   size_t start = A_rows_csr_h[row_ptr];
  //   size_t end = A_rows_csr_h[row_ptr + 1];
  //   size_t prev = RAND_MAX;
  //   for (size_t col_ptr = start; col_ptr < end; col_ptr++) {
  //     if (prev == A_cols_h[col_ptr]) {
  //       std::cout << "ERROR at row << "row_ptr: " << prev << " == " <<
  //       A_cols_h[col_ptr] << '\n';
  //     }
  //     prev = A_cols_h[col_ptr];
  //   }
  // }

  size_t *A_rows_coo_d;
  size_t *A_rows_csr_d;
  size_t *A_cols_d;

  CUDA_CHECK(hipMalloc(&A_rows_coo_d, nnz * sizeof(size_t)));
  CUDA_CHECK(hipMalloc(&A_cols_d, nnz * sizeof(size_t)));
  CUDA_CHECK(hipMalloc(&A_rows_csr_d, (rows + 1) * sizeof(size_t)));

  CUDA_CHECK(hipMemcpy(A_rows_coo_d, A_rows_coo_h.data(), nnz * sizeof(size_t),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(A_cols_d, A_cols_h.data(), nnz * sizeof(size_t),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(A_rows_csr_d, A_rows_csr_h.data(),
                        (rows + 1) * sizeof(size_t), hipMemcpyHostToDevice));

  GPUTimer timer;

  floatType *H_h, *HT_h, *res_h;
  CUDA_CHECK(hipHostMalloc(&H_h, rows * cols * sizeof(floatType)));
  CUDA_CHECK(hipHostMalloc(&HT_h, rows * cols * sizeof(floatType)));
  CUDA_CHECK(hipHostMalloc(&res_h, nnz * sizeof(floatType)));

  floatType *H_d, *HT_d, *res_d;
  CUDA_CHECK(hipMalloc(&H_d, rows * cols * sizeof(floatType)));
  CUDA_CHECK(hipMalloc(&HT_d, rows * cols * sizeof(floatType)));
  CUDA_CHECK(hipMalloc(&res_d, nnz * sizeof(floatType)));

  for (size_t i = 0; i < rows * cols; i++) {
    H_h[i] = (((floatType)rand()) / RAND_MAX - 0.5) * 100;
    HT_h[i] = H_h[i];
    if (print_debug) {
      if (i % cols == 0) {
        std::cout << "\n";
      }
      std::cout << H_h[i] << ' ';
    }
  }
  std::cout << "\n";
  // memset(H_h, 1, rows / 2 * cols * sizeof(floatType));
  // memset(H_h + rows / 2 * cols, 3, rows / 2 * cols * sizeof(floatType));
  memset(res_h, 0, nnz * sizeof(floatType));
  if (print_debug)
    std::cout << "Vector init complete\n";

  CUDA_CHECK(hipMemcpy(H_d, H_h, rows * cols * sizeof(floatType),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(HT_d, HT_h, rows * cols * sizeof(floatType),
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(res_d, res_h, nnz * sizeof(floatType),
                        hipMemcpyHostToDevice));

  const int numThreads = cols;

  double cpu_time = 0.0;
  if (checkRMSE) {
    for (int iter = 0; iter < runs; iter++) {
      timer.start();
      host_batched_dot(A_rows_coo_h.data(), A_cols_h.data(), nnz, cols, H_h,
                       res_h);
      cpu_time += timer.seconds() / runs;
    }
    if (print_debug) {
      std::cout << "Host: ";
      for (size_t iter = 0; iter < nnz; iter++) {
        std::cout << res_h[iter] << ' ';
      }
      std::cout << '\n';
    }
  }

  CUDA_CHECK(hipMemset(res_d, 0, nnz * sizeof(floatType)));

  // Parallelizes numBlocks_y over number of nnz up to max limit
  unsigned int numBlocks_y = (unsigned int)min(nnz, (size_t)MAX_GRID_Y);
  std::cout << "NumBlocks_y: " << numBlocks_y << '\n';

  double coo_time = 0.0;
  for (int iter = 0; iter < runs; iter++) {
    dotKernel2d_coo<<<{1, numBlocks_y}, {numThreads, 1}>>>(
        A_rows_coo_d, A_cols_d, nnz, cols, H_d, HT_d, res_d);
    timer.start();
    // dotKernel2d_coo<<<{1, numBlocks_y}, {numThreads, 1}>>>(
    //     A_rows_coo_d, A_cols_d, nnz, cols, H_d, HT_d, res_d);
    vanilla_attention_coo(A_rows_coo_d, A_cols_d, nnz, cols, H_d, HT_d, res_d);
    coo_time += timer.seconds() / runs;
  }

  if (print_debug) {
    std::cout << "COO: ";
    printVector(nnz, res_d);
  }

  if (checkRMSE) {
    auto myRMSE = rmse(nnz, res_h, res_d);
    printf("RMSE: %f\n", myRMSE);
  }

  CUDA_CHECK(hipMemset(res_d, 0, nnz * sizeof(floatType)));

  // Parallelizes numBlocks_y over the number of rows
  numBlocks_y = (unsigned int)min(rows, (size_t)MAX_GRID_Y);
  std::cout << "NumBlocks_y: " << numBlocks_y << '\n';
  assert(numBlocks_y <= MAX_GRID_Y &&
         "Matrix has too many rows for this implementations");
  // Parallelzes numBlocks_z over the number off nzz in cols
  double csr_time = 0.0;
  // for (int iter = 0; iter < runs; iter++) {
  //   dotKernel2d_csr<<<{1, numBlocks_y, numBlocks_y}, {numThreads, 1}>>>(
  //       A_rows_csr_d, A_cols_d, rows, cols, H_d, res_d);
  //   timer.start();
  //   dotKernel2d_csr<<<{1, numBlocks_y, numBlocks_y}, {numThreads, 1}>>>(
  //       A_rows_csr_d, A_cols_d, rows, cols, H_d, res_d);
  //   csr_time += timer.seconds() / runs;
  // }

  if (print_debug) {
    std::cout << "CSR: ";
    printVector(nnz, res_d);
  }

  if (checkRMSE) {
    auto myRMSE = rmse(nnz, res_h, res_d);
    printf("RMSE: %f\n", myRMSE);
  }

  hipsparseHandle_t cusparse_handle;
  CUSPARSE_CALL(hipsparseCreate(&cusparse_handle));
  // hipsparseSpMatDescr_t matA;
  // hipsparseDnMatDescr_t matB, matC;
  // void *dBuffer = NULL;
  // size_t bufferSize = 0;

  // CUSPARSE_CALL(hipsparseCreateCoo(&matA, rows, rows, nnz, A_rows_coo_d,
  //                                 A_cols_d, res_d, HIPSPARSE_INDEX_32I,
  //                                 HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));

  // double alpha = 1.0;
  // double beta = 0.0;

  printf("COO:     %.4f GB/s;\t%.4f GFLOPS (%f sec) \n", numBytes / coo_time,
         numFlops / coo_time, coo_time);
  printf("CSR:     %.4f GB/s;\t%.4f GFLOPS (%f sec) \n", numBytes / csr_time,
         numFlops / csr_time, csr_time);
  // printf("cusparse:  %.2f GB/s;  %.2f GFLOPS (%f sec) \n", numBytes /
  // cublas_time,
  //        numFlops / cublas_time, cublas_time);
  if (checkRMSE)
    printf("CPU:     %.4f GB/s;\t%.4f GFLOPS (%f sec) \n", numBytes / cpu_time,
           numFlops / cpu_time, cpu_time);

  hipsparseDestroy(cusparse_handle);
  CUDA_CHECK(hipHostFree(res_h));
  CUDA_CHECK(hipHostFree(H_h));
  CUDA_CHECK(hipHostFree(HT_h));

  CUDA_CHECK(hipFree(res_d));
  CUDA_CHECK(hipFree(H_d));
  CUDA_CHECK(hipFree(HT_d));
  CUDA_CHECK(hipFree(A_rows_csr_d));
  CUDA_CHECK(hipFree(A_rows_coo_d));
  CUDA_CHECK(hipFree(A_cols_d));
}
