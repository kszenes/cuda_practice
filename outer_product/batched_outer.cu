#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "timer.h"
#include <hipblas.h>
#include <vector>

#define CUDA_CHECK(val) {                                                  \
    if (val != hipSuccess) {                                              \
        printf("Error: %s in line %d\n",hipGetErrorName(val), __LINE__);  \
        exit(-1);                                                          \
    }                                                                      \
}

template <typename T>
void host_ger(const size_t, const size_t, const size_t, const T, const T*, const T*, T*);

template <typename T>
T rmse(const size_t, const T*, const T*);

template <typename T>
__global__
void device_ger(const size_t xDim, const size_t yDim, const T a, const T* x, const T* y, T* mat) {
    const int row = threadIdx.x + blockIdx.x * blockDim.x;
    const int col = threadIdx.y + blockIdx.y * blockDim.y;
    if (row < xDim && col < yDim) {
        mat[row + col * xDim] += a * x[row] * y[col];
    }
}

template <typename T>
__global__
void device_khatriRao(const size_t xDim, const size_t yDim, const size_t nBatches, const T a, const T* x, const T* y, T* mat) {
    const int row = threadIdx.x + blockIdx.x * blockDim.x;
    const int col = threadIdx.y + blockIdx.y * blockDim.y;
    const int batch = threadIdx.z + blockIdx.z * blockDim.z;
    if (row < xDim && col < yDim && batch < nBatches) {
        mat[(batch * xDim * yDim) + row + col * xDim] += a * x[(batch * xDim) + row] * y[(batch * yDim) + col];
    }
}

// template <typename T>

int main() {
    const int runs = 1;
    const bool checkRMSE = false;
    #define FLOAT

    #if defined(DOUBLE)
        #define DTYPE double
    #elif defined(FLOAT)
        #define DTYPE float
    #endif

    CUDA_CHECK(hipSetDevice(1));


    const DTYPE a = 1.f;
    // const DTYPE b = 2.9f;
    const size_t size = 1 << 10;
    const size_t xDim = size;
    const size_t yDim = size;
    const size_t nBatches = size;

    const double numFlops = 3 * xDim * yDim * 1e-9;
    const double numBytes = (2 * xDim * yDim + xDim + yDim) * nBatches * sizeof(DTYPE) * 1e-9;
    printf("Runs: %d\n", runs);
    printf("xDim: %zu\nyDim: %zu\n", xDim, yDim);
    printf("Memory usage = %f GB\n", numBytes);

    GPUTimer timer;

    DTYPE *x_h, *y_h, *mat_h;

    CUDA_CHECK(hipHostMalloc(&mat_h, nBatches * xDim * yDim * sizeof(DTYPE)))
    CUDA_CHECK(hipHostMalloc(&x_h, nBatches * xDim * sizeof(DTYPE)));
    CUDA_CHECK(hipHostMalloc(&y_h, nBatches * yDim * sizeof(DTYPE)));


    DTYPE *x_d, *y_d, *mat_d;
    CUDA_CHECK(hipMalloc(&mat_d, nBatches * xDim * yDim * sizeof(DTYPE)));
    CUDA_CHECK(hipMalloc(&x_d, nBatches * xDim * sizeof(DTYPE)));
    CUDA_CHECK(hipMalloc(&y_d, nBatches * yDim * sizeof(DTYPE)));

    if (checkRMSE) {
        for (size_t i = 0; i < nBatches * xDim; i++) {
            x_h[i] = (((DTYPE) rand()) / RAND_MAX - 0.5) * 100;
        }
        for (size_t i = 0; i < nBatches * yDim; i++) {
            y_h[i] = (((DTYPE) rand()) / RAND_MAX - 0.5) * 100;
        }
        
        for (size_t i = 0; i <  nBatches * xDim*yDim; i++){
            mat_h[i] = 0.0;
        }
        CUDA_CHECK(hipMemcpy(x_d, x_h, nBatches * xDim * sizeof(DTYPE), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(y_d, y_h, nBatches * yDim * sizeof(DTYPE), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(mat_d, mat_h, nBatches * xDim * yDim * sizeof(DTYPE), hipMemcpyHostToDevice));
    }

    dim3 numThreads(32, 32);
    dim3 numBlocks(
        (xDim + numThreads.x - 1) / numThreads.x,
        (yDim + numThreads.y - 1) / numThreads.y
    );

    double cpu_time = 0.0;
    if (checkRMSE) {
        for (int iter = 0; iter < runs; iter++) {
            if (!checkRMSE) host_ger(xDim, yDim, 1, a, x_h, y_h, mat_h);
            timer.start();
            host_ger(xDim, yDim, nBatches, a, x_h, y_h, mat_h);
            cpu_time += timer.seconds() / runs;
        }
    }

    DTYPE *tmp_d;
    CUDA_CHECK(hipMalloc(&tmp_d, nBatches * xDim * yDim * sizeof(DTYPE)));
    CUDA_CHECK(hipMemset(tmp_d, 0, nBatches * xDim * yDim * sizeof(DTYPE)));

    // double io_time = 0.0;
    // for (int iter = 0; iter < runs; iter++) {
    //     timer.start();
    //     device_io<<<numBlocks, numThreads>>>(n, a, x_d, tmp_d);
    //     io_time += timer.seconds() / runs;
    // }

    double gpu_time = 0.0;
    for (int iter = 0; iter < runs; iter++) {
        if (!checkRMSE) device_ger<<<numBlocks, numThreads>>>(xDim, yDim, a, x_d, y_d, tmp_d);
        timer.start();
        for (size_t batch = 0; batch < nBatches; batch++) {
            device_ger<<<numBlocks, numThreads>>>(xDim, yDim, a, x_d + batch * xDim, y_d + batch * yDim, tmp_d + batch * xDim * yDim);
            // device_ger<<<numBlocks, numThreads>>>(xDim, yDim, a, x_d, y_d, tmp_d);
        }
        gpu_time += timer.seconds() / runs;
    }
    if (checkRMSE) {
        auto myRMSE = rmse(xDim * yDim, mat_h, tmp_d);
        printf("RMSE Batched: %f\n", myRMSE);
    }

    numThreads = dim3(16, 16, 4);
    numBlocks = dim3(
        (xDim + numThreads.x - 1) / numThreads.x,
        (yDim + numThreads.y - 1) / numThreads.y,
        (nBatches + numThreads.z - 1) / numThreads.z
    );

    CUDA_CHECK(hipMemset(tmp_d, 0, nBatches * xDim * yDim * sizeof(DTYPE)));
    double gpu_batched_time = 0.0;
    for (int iter = 0; iter < runs; iter++) {
        if (!checkRMSE) device_ger<<<numBlocks, numThreads>>>(xDim, yDim, a, x_d, y_d, tmp_d);
        timer.start();
        device_khatriRao<<<numBlocks, numThreads>>>(xDim, yDim, nBatches, a, x_d, y_d, tmp_d);
        gpu_batched_time += timer.seconds() / runs;
    }
    if (checkRMSE) {
        auto myRMSE = rmse(xDim * yDim, mat_h, tmp_d);
        printf("RMSE Tiled: %f\n", myRMSE);
    }

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasStatus_t error;
    CUDA_CHECK(hipMemset(tmp_d, 0, nBatches * xDim * yDim * sizeof(DTYPE)));
    double cublas_time = 0.0;
    for (int iter = 0; iter < runs; iter++) {
        if (!checkRMSE) {
            #if defined(DOUBLE)
            error = hipblasDger(
                handle, xDim, yDim, &a, x_d, 1, y_d, 1, tmp_d, xDim
            );
            #elif defined(FLOAT)
            error = hipblasSger(
                handle, xDim, yDim, &a, x_d, 1, y_d, 1, tmp_d, xDim
            );
            #endif
        }
        timer.start();
        for (size_t batch = 0; batch < nBatches; batch++) {
            #if defined(DOUBLE)
            error = hipblasDger(
                handle, xDim, yDim, &a, x_d + batch * xDim, 1, y_d + batch * yDim, 1, tmp_d + batch * xDim * yDim, xDim
            );
            #elif defined(FLOAT)
            error = hipblasSger(
                handle, xDim, yDim, &a, x_d + batch * xDim, 1, y_d + batch * yDim, 1, tmp_d + batch * xDim * yDim, xDim
            );
            #endif
        }
        cublas_time += timer.seconds() / runs;

        if (error != HIPBLAS_STATUS_SUCCESS) {
            printf("Error\n");
            exit(-1);
        }
    }

    if (checkRMSE) {
        auto myRMSE = rmse(xDim * yDim, mat_h, tmp_d);
        printf("RMSE cuBLAS: %f\n", myRMSE);
    }
    // printf("IO: %f sec\n", io_time);
    printf("Batched:\t%.2f GB/s; %.2f GFLOPS (%f sec) \n", numBytes/gpu_time, numFlops/gpu_time, gpu_time);
    printf("Tiled:  \t%.2f GB/s; %.2f GFLOPS (%f sec) \n", numBytes/gpu_batched_time, numFlops/gpu_batched_time, gpu_batched_time);
    printf("cuBLAS: \t%.2f GB/s; %.2f GFLOPS (%f sec) \n", numBytes/cublas_time, numFlops/cublas_time, cublas_time);
    if (checkRMSE) printf("CPU:\t\t%.2f GB/s; %.2f GFLOPS (%f sec) \n", numBytes/cpu_time, numFlops/cpu_time, cpu_time);

    hipblasDestroy(handle);
    CUDA_CHECK(hipHostFree(mat_h));
    CUDA_CHECK(hipHostFree(x_h));
    CUDA_CHECK(hipHostFree(y_h));

    CUDA_CHECK(hipFree(mat_d));
    CUDA_CHECK(hipFree(tmp_d));
    CUDA_CHECK(hipFree(x_d));
    CUDA_CHECK(hipFree(y_d));
}


template <typename T>
void host_ger(const size_t xDim, const size_t yDim, const size_t nBatches, const T a, const T* x_h, const T* y_h, T* mat_h) {
    for (size_t batch = 0; batch < nBatches; batch++) {
        for (size_t row = 0; row < xDim; row++) {
            for (size_t col = 0; col < yDim; col++) {
                mat_h[(batch * xDim * yDim) + row + xDim * col] = a * x_h[(batch * xDim) + row] * y_h[(batch * yDim) + col];
            }
        }

    }
}

template <typename T>
T rmse(const size_t n, const T* v_ref, const T* v_d) {
    T *v_h = (T*) malloc(n * sizeof(T));
    CUDA_CHECK(hipMemcpy(v_h, v_d, n * sizeof(T), hipMemcpyDeviceToHost));

    T diff = 0.0f;
    for (size_t i = 0; i < n; i++) {
        // printf("Ref: %f; Dev: %f\n", v_ref[i], v_h[i]);
        diff += std::sqrt((v_ref[i] - v_h[i]) * (v_ref[i] - v_h[i]));
        // printf("mat[%zu] = %f\n", i, v_h[i]);
    }

    free(v_h);
    return diff / n;

}
